#include "hip/hip_runtime.h"
#include "BigNum.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <unistd.h>
#define SIZE ((5))

/*
__device__ inline void  cuda_printNum(unsigned int *__restrict__ num){
	unsigned int i;
	for(i = 0; i < SIZE; i ++)
		printf("%x ", num[i]);
	printf("\n");
}*/

__device__ inline void  cuda_setZero(unsigned int * num){
	unsigned int i;
    #pragma unroll
	for(i = 0; i < SIZE; i ++)
		num[i] = 0;
}

__device__ inline void  cuda_addNum(unsigned int  * A, const unsigned int * B){
	unsigned int i;
	unsigned long int tmp;
	unsigned long int tmp_carry;
	unsigned int carry[SIZE];

    
    #pragma unroll
    for(i = 0; i < SIZE - 1; i ++){
		tmp = ((unsigned long int) A[i] + B[i]);
		//A[i] = (tmp & 0xFFFFFFFF) + carry;
		A[i] = (tmp & 0xFFFFFFFF);
        tmp_carry = ((unsigned long int) tmp & 0xFFFFFFFF00000000); 
		carry[i] = ((unsigned long int) tmp_carry >> 32);
	}
    A[i] = ((unsigned long int) A[i] + B[i]);
    
    #pragma unroll
    for(i = 1; i < SIZE; i ++){
        A[i] = A[i] + carry[i - 1];
    }

    /*
    for(i = 0; i < SIZE - 1; i ++){
        tmp = ((unsigned long int) A[i] + B[i]);
        A[i] = (tmp & 0xFFFFFFFF) + carry;
        tmp_carry = ((unsigned long int) tmp & 0xFFFFFFFF00000000); 
        carry = ((unsigned long int) tmp_carry >> 32);
    }
    */
}

__device__ inline void  cuda_addfromto(unsigned int * A, const unsigned int * B, const unsigned int from){
    unsigned int i;
    unsigned long int tmp;
    unsigned long int tmp_carry;
    unsigned int carry[SIZE];
    
    #pragma unroll
    for(i = from; i < SIZE - 1; i ++){
        tmp = ((unsigned long int) A[i] + B[i-from]);
        //A[i] = (tmp & 0xFFFFFFFF) + carry;
        A[i] = (tmp & 0xFFFFFFFF);
        tmp_carry = ((unsigned long int) tmp & 0xFFFFFFFF00000000); 
        carry[i] = ((unsigned long int) tmp_carry >> 32);
    }
    A[SIZE - 1] = ((unsigned long int) A[SIZE - 1] + B[SIZE - 1]);
    #pragma unroll
    for(i = from + 1; i < SIZE; i ++){
        A[i] = A[i] + carry[i - 1];
    }

}
__device__ inline void  cuda_addOne(unsigned int * A, unsigned int * C){ // not used for anything currently
	unsigned int i;
	unsigned long int tmp;
	unsigned long int tmp_carry;
	unsigned int carry = 0;

	for(i = 0; i < SIZE; i ++){
		if ( i == 0)
			tmp = ((unsigned long int) A[i] + 1);
		else
			tmp = ((unsigned long int) A[i]);	
		C[i] = (tmp & 0xFFFFFFFF) + carry;
		tmp_carry = ((unsigned long int) tmp & 0xFFFFFFFF00000000); 
		carry = ((unsigned long int) tmp_carry >> 32);
	}
}
__device__ inline void  cuda_subNum(unsigned int * A, const unsigned int * B){
	unsigned int i;
	unsigned long int tmp;
	unsigned int carry[SIZE];
    
    #pragma unroll
    for(i = 0; i < SIZE; i ++){
		tmp = ((unsigned long int) 0xFFFFFFFF00000000 + A[i]);
		tmp = ((unsigned long int) tmp - B[i]);
		carry[i] = (0xFFFFFFFF - ((unsigned long int) ((unsigned long int) 0xFFFFFFFF00000000 & tmp) >> 32));
		A[i] = (tmp & 0xFFFFFFFF);
	}

    #pragma unroll
    for(i = 1; i < SIZE; i ++){
        A[i] = A[i] - carry[i - 1];
    }    
}

__device__ inline void  cuda_copyNum(unsigned int * A, const unsigned int * B){
	unsigned int i;
	#pragma unroll
    for(i = 0; i < SIZE; i ++)
		A[i] = B[i];
}

__device__ inline void  cuda_copyFromTo(unsigned int * A, const unsigned int * B, unsigned int from){
	unsigned int i;
	for(i = from; i < SIZE; i ++)
		A[i] = B[i - from];
}

__device__ inline void  cuda_shiftLeftNum(unsigned int * A){ // shift for 32 bits in one step would be nice to consider
    char flg;
    char flc = 0;
    for (unsigned int j = 0 ; j < SIZE; j++){
    	((((unsigned int) A[j] & 0x80000000)) ? flg = 1 : flg = 0 );
    	A[j] = (A[j] << 1);
    	if(flc)
    		A[j] |= 1;
    	(flg == 1 ? flc = 1 : flc = 0);
    }
}
__device__ inline void  cuda_shiftRightNum(unsigned int * A){
    char flg;
    char flc = 0;
    for (unsigned char j = SIZE - 1; ; j --){
        ((((unsigned int) A[j] & 0x00000001)) ? flg = 1 : flg = 0 );
        A[j] = (A[j] >> 1);
        if(flc)
            A[j] |= 0x80000000;
        (flg == 1 ? flc = 1 : flc = 0);
        if (j == 0)
            break;
    }
}

__device__ inline void  cuda_mulNum(unsigned int * A, const unsigned int * B){
    //unsigned int * tmpa = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpa[SIZE];
    //unsigned int * tmpc = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpc[SIZE];
    cuda_copyNum(tmpa, A);
    cuda_setZero(tmpc);
    unsigned long int j = 1;
    for(unsigned int k = 0; k < 32; k++){ // needed to be changed accordingly with sizeof the datatype
        for(unsigned int i = 0; i < SIZE; i++){
            if (B[i] & j){
                cuda_addfromto(tmpc, tmpa, i);
            }
        }
        j <<= 1;
        cuda_shiftLeftNum(tmpa);
    }
    cuda_copyNum(A, tmpc);
    //free(tmpa);
    //free(tmpc);
}

__device__ inline void  cuda_powNum(unsigned int * A){
    //unsigned int * tmpa = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpa[SIZE];
    //unsigned int * tmpc = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpc[SIZE];
    cuda_copyNum(tmpa, A);
    cuda_setZero(tmpc); 
    unsigned long int j = 1;
    for(unsigned int k = 0; k < 32; k++){ // needed to be changed accordingly with sizeof the datatype
        for(unsigned int i = 0; i < SIZE; i++){
            if (A[i] & j){
                cuda_addfromto(tmpc, tmpa, i);
            }
        }
        j <<= 1;
        cuda_shiftLeftNum(tmpa);
    }
    cuda_copyNum(A, tmpc);
    //free(tmpa);
    //free(tmpc);
}

__device__ inline char cuda_zeroNum(const unsigned int * num){
	unsigned int i;
	for(i = 0; i < SIZE; i ++)
		if (num[i] > 0)
			return 0;
	return 1;
}

__device__ inline char cuda_bigger(const unsigned int * A, const unsigned int * B){
	unsigned int i;
	for(i = SIZE - 1; ; i --){
		if (A[i] > B[i])
			return 1;
		if (A[i] < B[i])
			return 0;
		if (i == 0)
			break;
	}
	return 2;
}

__device__ inline unsigned int cuda_getHighBit(unsigned int * A){
    unsigned int bits_size = SIZE * 32;
    unsigned int l = 0;
    unsigned int tmp;
    for (long int i = SIZE - 1; i != -1; i --){
    	tmp = 0x80000000;
        while ((A[i] & tmp) == 0 && tmp != 0){
            tmp >>= 1;
            l ++;
        }
        if(tmp != 0)
            break;
    }
    l = bits_size - l;
    return l;
}


__device__ inline char cuda_isEven(const unsigned int * A){
	if ((A[0] & 1) == 0)
		return 1;
	return 0;
}
__device__ inline char cuda_isOne(const unsigned int * A){
	for (unsigned int i = 1; i < SIZE; i++){
		if (A[i] > 0)
			return 0;
	}
	if (A[0] == 1)
		return 1;
	return 0;
}

__device__ inline void  cuda_getNum(unsigned int a, unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    A[where] = 1;
    while (which){
        A[where] <<= 1;
        which --;
    }
}

__device__ inline unsigned int cuda_getBit(unsigned int a, const unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    unsigned int pnt = ((unsigned int)1 << which);
    return ((unsigned int) A[where] & pnt); 
}
__device__ inline void  cuda_setBit(unsigned int a, unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    unsigned int pnt = ((unsigned int) 1 << which);
    A[where] |= pnt; 
}
__device__ inline void  cuda_unsBit(unsigned int a, unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    unsigned int pnt = ((unsigned int) 1 << which);
    A[where] &= ~(pnt); 
}


__device__ inline void  cuda_divNum(unsigned int * N, unsigned int * D, unsigned int * Q){
    if (cuda_zeroNum(D) || cuda_zeroNum(N) || cuda_bigger(D, N) == 1){
        return;
    }
    //unsigned int * R = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int R[SIZE];
    cuda_setZero(Q);
    cuda_setZero(R);
    unsigned int i = SIZE*32-1;
    while (1){ // 32 is sizeof int, need to changed to sizeof
        cuda_shiftLeftNum(R);
        if (cuda_getBit(i, N)){
            R[0] |= 1;
        }else{
            R[0] &= ~((unsigned int) 1);
        }
        if(cuda_bigger(R, D) == 2 || cuda_bigger(R, D) == 1){
            cuda_subNum(R, D);
            cuda_setBit(i, Q);
        }
        if (i == 0)
            break;
        i --;
    }
    cuda_copyNum(N, R);
    //free(R);
}

__device__ inline void  cuda_shiftLeftNumBy(unsigned int * A, unsigned int gen){ // shift for 32 bits in one step would be nice to consider
    unsigned int howmuch = gen / 32; // needed to be changed accordingly to datatype
	unsigned int wheremc = gen % 32; // needed to be changed accordingly to datatype
	//printf("%u \n", howmuch);
	if(howmuch > 0){
		for(unsigned int k = SIZE - 1;  ; k --){
			A[k] = A[k - howmuch];
			if (k - howmuch == 0)
				break;
		}
    }
	//printf("%u \n", wheremc);
	while(wheremc){
    	cuda_shiftLeftNum(A);
    	wheremc --;
    }
    /*while(gen){
    	shiftLeftNum(A);
    	gen --;
    }*/
    /*char flg;
    char flc = 0;
    for (unsigned int j = 0 ; j < size; j++){
    	((((unsigned int) A[j] & 0x80000000)) ? flg = 1 : flg = 0 );
    	A[j] = (A[j] << 1);
    	if(flc)
    		A[j] |= 1;
    	(flg == 1 ? flc = 1 : flc = 0);
    }*/
}

/*
__device__ void  cuda_modNum(unsigned int * A, unsigned int * B){
    if (cuda_zeroNum(A) || cuda_zeroNum(B) || cuda_bigger(B, A) == 1){
        return;
    }
    //unsigned int * tmp_b = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmp_b[SIZE];
    char gr_des;
    unsigned int dr;
    unsigned int gen;
    while( 1 ){
        cuda_copyNum(tmp_b, B);
        dr = 0;
        //gr_des = bigger(tmp_b, A);
        gen = cuda_getHighBit(A) - cuda_getHighBit(tmp_b);
        cuda_shiftLeftNumBy(tmp_b, gen);
        gr_des = cuda_bigger(tmp_b, A);
        // pokud bylo rovno ok
        // pokud bylo větší shift doprava
        if(gr_des == 1){
            cuda_shiftRightNum(tmp_b);
            dr --;
        }
        cuda_subNum(A, tmp_b); // residuo
        if(cuda_bigger(B, A)){
            break;
        }
    }
    //free(tmp_b);
}
*/

__device__ inline void  cuda_modNum(unsigned int * A, const unsigned int * B){
    if (cuda_zeroNum(A) || cuda_zeroNum(B) || cuda_bigger(B, A) == 1){
        return;
    }
    unsigned int tmp_b[SIZE];
    while( 1 ){
        cuda_copyNum(tmp_b, B);
        unsigned int dr = 0;
        char gr_des = cuda_bigger(tmp_b, A);
        while(gr_des == 0){ // dokud je B menší než A
            // shiftuj doleva
            cuda_shiftLeftNum(tmp_b);
            gr_des = cuda_bigger(tmp_b, A);
            dr ++;
        }
        // pokud bylo rovno ok
        // pokud bylo větší shift doprava
        if(gr_des == 1){
            cuda_shiftRightNum(tmp_b);
            dr --;
        }
        cuda_subNum(A, tmp_b); // residuo
        if(cuda_bigger(B, A)){
            break;
        }
    }
    //free(tmp_b);
}



__device__  inline void cuda_gcd(unsigned int * A, unsigned int * B){
    
    /*unsigned int R[SIZE];
    while(!cuda_zeroNum(B)){
        cuda_copyNum(R, A);
        cuda_modNum(R, B);
        cuda_copyNum(A, B);
        cuda_copyNum(B, R);
    }*/
    unsigned int t [SIZE];
    unsigned int shift;

    if(cuda_zeroNum(B)){
        return;      
    }
    if(cuda_zeroNum(A)){
        cuda_copyNum(A, B);
        return;
    }
    for(shift = 0; ((A[0] | B[0]) & 1) == 0; ++ shift){
        cuda_shiftRightNum(A);
        cuda_shiftRightNum(B);
    }
    while((A[0] & 1) == 0){
        cuda_shiftRightNum(A);
    }
    do{
        while((B[0] & 1) == 0){
            cuda_shiftRightNum(B);
        }
        if(cuda_bigger(A, B) == 1){
            cuda_copyNum(t, B);
            cuda_copyNum(B, A);
            cuda_copyNum(A, t);
        }
        cuda_subNum(B, A);
    } while (! cuda_zeroNum(B));
    cuda_shiftLeftNumBy(A, shift);
}

__device__  inline void cuda_fxfun(const unsigned int * N, unsigned int * X, unsigned int * C){
    //setZero(Y);
    cuda_powNum(X);
    cuda_modNum(X, N);
    cuda_addNum(X, C);
    cuda_modNum(X, N);
    //copyNum(Y, X);
}
//__global__ inline void pollardKernel(unsigned int * N, unsigned int * mem_xyc, unsigned int * result, unsigned int * dbgs){
__global__ void pollardKernel(const unsigned int * N, unsigned int * result){
    unsigned int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int X[SIZE];
    unsigned int Y[SIZE];
    unsigned int C[SIZE];
    unsigned int G[SIZE];
    unsigned int N_tmp[SIZE];
    unsigned int abs_mxy[SIZE];
    
    cuda_setZero(X);
    X[0] = 0x07;
    cuda_setZero(C);
    C[0] = threadID + 1;
    cuda_setZero(G);
    G[0] = 0x01;
    cuda_copyNum(Y, X);
    cuda_fxfun(N, Y, C);
   
    unsigned int check = 0;

    while (cuda_isOne(G)){
        cuda_fxfun(N, X, C);
        cuda_fxfun(N, Y, C);
        cuda_fxfun(N, Y, C);
        if(cuda_bigger(X, Y) == 1){
            cuda_copyNum(abs_mxy, X);
            cuda_subNum(abs_mxy, Y);
        }else{
            cuda_copyNum(abs_mxy, Y);
            cuda_subNum(abs_mxy, X);    
        }
        cuda_copyNum(G, abs_mxy);
        cuda_copyNum(N_tmp, N);
        cuda_gcd(G, N_tmp);
        check ++;
        if ((check % 1000 == 0) && !cuda_zeroNum(result)){
            return;
        }
    }
    
    cuda_copyNum(result, G);    
}
