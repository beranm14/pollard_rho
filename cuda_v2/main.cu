#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h> 
#include "BigNum/BigNum.h"
#include "BigNum/BigNum.cuh"
#define SIZE ((5))

//

/*
static long gcdl(long a, long b) {
  long r;
  while(b!=0){
    r = a % b;
    a = b;
    b = r;
		printf("**************************\n");
		printf("N:\n");
		printf("%ld \n", a);    
		printf("M:\n");
		printf("%ld \n", b);    
    getchar(); 
  }
  return a;
}*/


void getGpuNfo(){
	// run some tests on CUDA device
    int num;
    if (hipGetDeviceCount(&num)){
    	printf("Cannot get number of CUDA devices\n");
    	return;
	}
	if (num < 1){
		printf("No CUDA devices found\n");
    	return;
	}
    hipDeviceProp_t prop;
    int   MaxDevice = -1;
    int   MaxGflops = -1;
    for (int dev=0;dev<num;dev++)
    {
        if (hipGetDeviceProperties(&prop,dev)){
        	printf("Error getting device %d properties\n",dev);
        	return;	
    	}
    	int Gflops = prop.multiProcessorCount * prop.clockRate;
        printf("CUDA Device %d: %s Gflops %f Processors %d Threads/Block %d\n",dev,prop.name,1e-6*Gflops,prop.multiProcessorCount,prop.maxThreadsPerBlock);
        if(Gflops > MaxGflops)
        {
            MaxGflops = Gflops;
            MaxDevice = dev;
        }
    }
    
    //  Print and set device
    if (hipGetDeviceProperties(&prop,MaxDevice)){
    	printf("Error getting device %d properties\n", MaxDevice);
    	return;
    }
    printf("Fastest CUDA Device %d: %s\n",MaxDevice,prop.name);
    // hipSetDevice(MaxDevice);    
    //hipSetDevice(0);
    //  Return max thread count
	printf("maxThreadsPerBlock %d \n", prop.maxThreadsPerBlock);
	return;
}


void genNum(unsigned int * x){
	setZero(x);
	x[0] = 0x07;
}

void genC(unsigned int * x, unsigned int * counter){
	setZero(x);
	copyNum(x, counter);
	addOne(counter);
}

void printMat(unsigned int * mem_xyc, unsigned int blocks, unsigned int threads){
	for(unsigned int i = 0; i < 3 * blocks * threads; i += 3){
		printNum ( mem_xyc + SIZE * ( i     )); // X
		printNum ( mem_xyc + SIZE * ( i + 1 )); // Y
		printNum ( mem_xyc + SIZE * ( i + 2 )); // C 
		printf("===================================================== \n");
	}
}

bool compareFields(unsigned int * a, unsigned int * b, unsigned int count){
	for (unsigned int i = 0; i < count; ++i){
		if (a [i] != b[i])
			return false;
	}
	return true;
}

void PollardRhoCu(unsigned int * N, unsigned int blocks, unsigned int threads){
	/*
The idea I've got in mind is to prepare multiple starting points
for X, Y and C
and each thread can actually count Y in place without bothering CPU
Therefore reasonable aproach would be to just prepare huge chunk of memory
for GPU to work with.

This is going to be changed. Let's make just one kernel which is going to start
on it's own given X, Y, C. All of the threads are going to iterate in algorithm
as it is and check in memory array if anyone actually found sthing.
If so all the threads just stop leaving result for the CPU to copy and show to user.
	*/
	if (isEven(N)){
		zeroNum(N);
		N[0] = 2;
		return;
	}
	
	// unsigned int sz = 3 * blocks * threads * SIZE;
	unsigned int * result = (unsigned int *) malloc(sizeof(unsigned int) * SIZE);
	setZero(result);
	
	unsigned int * gpu_N;
  	hipMalloc((void **)&gpu_N, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_N, N, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

  	unsigned int * gpu_result;
  	hipMalloc((void **)&gpu_result, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_result, result, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

  	hipFuncSetCacheConfig(reinterpret_cast<const void*>(pollardKernel), hipFuncCachePreferL1);
	
	printf("Running Kernels\n");
	
	pollardKernel<<<blocks, threads>>>(gpu_N, gpu_result);
	//SinglePollardKernel(N, result);

	hipDeviceSynchronize();
	hipMemcpy(result, gpu_result, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	copyNum(N, result);
	hipFree(gpu_result);
	hipFree(gpu_N); 
}


int main(int argc, char **argv) {
	unsigned int blocks = 0;
	unsigned int threads = 0;
	unsigned int debug = 0;
	if(argc == 4){
		debug = atoi(argv[1]);
		blocks = atoi(argv[2]);
		threads = atoi(argv[3]);
	}
	hipSetDevice(0);
	//getGpuNfo();
	unsigned int N[SIZE];
	setZero(N);
    //N[8] = 1;
    //N[8] = 1;
    //N[0] = 1;
    //N[1] = 0x00000111;
    //N[0] = 0x10010009;
    //N[0] = 0x00044d69;
    //123432322333420120051
    //N[0] = 0x20120051;
    //N[1] = 0x23223334;
    //N[3] = 0x00012343;
    // cb3b6f92b9d54fd
    //N[1] = 0x0cb3b6f9;
    //N[0] = 0x2b9d54fd;
    //a8bc39b45
    //N[1] = 0x0000000a;
    //N[0] = 0x8bc39b45;
    //
    //N[0] = 0x00000121;
    //fd42d4eb2c4b7b1
	
    if(debug == 1){
    	float milliseconds = 0;
    	hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

    	N[1] = 0x0fd42d4e;
    	N[0] = 0xb2c4b7b1;

    	hipEventRecord(start, 0);

    	PollardRhoCu(N, blocks, threads);
    	
    	hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds, start, stop);
		
		printf("Results \n");
   		printNum(N);
		printf("Time: ");
		printf("%f\n", milliseconds);
		printf("********************\n");
    }else{
    	// custom
    	//fd42d4eb2c4b7b1 * 3fa81e47
    	N[1] = 0x0fd42d4e;
    	N[0] = 0xb2c4b7b1;
    	PollardRhoCu(N, 128, 128);
		printf("Results \n");
   		printNum(N);
		printf("********************\n");
	}
    /*unsigned int X[32];
	unsigned int C[32];
	unsigned int X[32];
	unsigned int Y[32];
	unsigned int C[32];
	unsigned int N[32];*/
	//void fxfun(unsigned int * N, unsigned int * X, unsigned int * C, unsigned int * Y)
	//setZero(N, 32);	
	//setZero(R, 32);
	//N[0] = 0x00000009;
	//0xF00000007530eca9
    /*N[3] = 0x8ebaf1b9;
    N[2] = 0xfebaf1ba;
    N[1] = 0xfeba9876;
    N[0] = 0x7530eca9;*/
    /*0 x 1 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000001*/
    /*N[8] = 1;
    N[0] = 1;
    PollardRho(N, R, 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(R, 32);
	printf("********************\n");
	*//*	
	setZero(A, 32);
	setZero(C, 32);
	setZero(N, 32);	
	setZero(X, 32);	
	setZero(Y, 32);	
	//N[1] = 0xF0010999;
    N[0] = 0x12345678;
    C[1] = 0x00000001;
	X[15] = 0x00000001;
	X[0] = 0x10000002;
	fxfun(N, X, C, Y , 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(X, 32);
    printNum(C, 32);
	printNum(Y, 32);	*/
	/*unsigned int A[32];
	unsigned int B[32];
	printf("********************\n"); 	
	setZero(A, 32);
	setZero(B, 32);
	//1B6 9B4B ACD0 5F15	
	//A[1] = 0x01B69B4B;
    //A[0] = 0x00000000;
	A[31] = 0x80000999; // to keep
    A[0] = 0x10000002; // to keep
	//A[0] = 0x80000003;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	//B[1] = 0x00000001;
	//B[0] = 0x00000000;
	//B[1] = 0x80000999; // to keep
	B[0] = 0x10001010; // to keep
	//gcd(A, B, 32);
    modNum(A, B, 32);
    printf("Results \n");
    printNum(A, 32);*/
	
	//printNum(B, 32);
/*
	printf("********************\n");
    unsigned int A[2*32];
	unsigned int B[2*32];
	setZero(A, 2*32);
	setZero(B, 2*32);
	A[0] = 1;    
	copyFromTo(B, A, 32, 2*32);
	printNum(B, 2*32);
*/
   /* printf("********************\n");
    unsigned int N[32];
	setZero(N, 32);
    unsigned int D[32];
	setZero(D, 32);
    unsigned int Q[32];
	setZero(Q, 32);
    
    //N[1] = 0x00000111;
    N[0] = 0x00000008;
    //D[1] = 0x00000111;
    D[0] = 0x00000002;
    
    divNum(N, D, Q, 32);
    printf("Results\n");
    printNum(N, 32);
    printNum(D, 32);
    printNum(Q, 32);
    printf("********************\n");
	return 0;*/
	/*
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[1] = 0xF0010999;
    A[0] = 0x12345678;
    B[1] = 0xF0000000;
	B[0] = 0x7530eca9;
	gcd(A, B, 32);
    //modNum(A, B, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(R, 32);
	printf("********************\n");*/
	/*setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	//1B6 9B4B ACD0 5F15
	A[0] = 0x00000008;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	B[0] = 0x00000008;
	divNum(A, B, D, R, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[0] = 0x89;
    //DB4 DA5F 7EF4 12B1
	B[0] = 0x21;
	gcd(A, B, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);*/
	//printf("gcdl %ld\n",gcdl(0x00000005, 0x00000002));
	hipDeviceReset();
	return 0;
}
