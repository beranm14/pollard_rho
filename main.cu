#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h> 
#include "BigNum/BigNum.h"
//#include "BigNum/BigNum.cuh"
#define SIZE ((32))

// =======================================

__device__ void  cuda_printNum(unsigned int * num){
	unsigned int i;
	for(i = 0; i < SIZE; i ++)
		printf("%x ", num[i]);
	printf("\n");
}

__device__ void  cuda_setZero(unsigned int * num){
	unsigned int i;
	for(i = 0; i < SIZE; i ++)
		num[i] = 0;
}

__device__ void  cuda_addNum(unsigned int * A, unsigned int * B){
	unsigned int i;
	unsigned long int tmp;
	unsigned long int tmp_carry;
	unsigned int carry = 0;
	for(i = 0; i < SIZE; i ++){
		tmp = ((unsigned long int) A[i] + B[i]);
		A[i] = (tmp & 0xFFFFFFFF) + carry;
		tmp_carry = ((unsigned long int) tmp & 0xFFFFFFFF00000000); 
		carry = ((unsigned long int) tmp_carry >> 32);
	}
}
__device__ void  cuda_addOne(unsigned int * A, unsigned int * C){
	unsigned int i;
	unsigned long int tmp;
	unsigned long int tmp_carry;
	unsigned int carry = 0;
	for(i = 0; i < SIZE; i ++){
		if ( i == 0)
			tmp = ((unsigned long int) A[i] + 1);
		else
			tmp = ((unsigned long int) A[i]);	
		C[i] = (tmp & 0xFFFFFFFF) + carry;
		tmp_carry = ((unsigned long int) tmp & 0xFFFFFFFF00000000); 
		carry = ((unsigned long int) tmp_carry >> 32);
	}
}
__device__ void  cuda_subNum(unsigned int * A, unsigned int * B){
	unsigned int i;
	unsigned long int tmp;
	unsigned int carry = 0;
	for(i = 0; i < SIZE; i ++){
		tmp = ((unsigned long int) 0xFFFFFFFF00000000 + A[i]);
		tmp = ((unsigned long int) tmp - (B[i] + carry));
		carry = (0xFFFFFFFF - ((unsigned long int) ((unsigned long int) 0xFFFFFFFF00000000 & tmp) >> 32));
		A[i] = (tmp & 0xFFFFFFFF);
	}
}

__device__ void  cuda_copyNum(unsigned int * A, unsigned int * B){
	unsigned int i;
	for(i = 0; i < SIZE; i ++)
		A[i] = B[i];
}

__device__ void  cuda_copyFromTo(unsigned int * A, unsigned int * B, unsigned int from){
	unsigned int i;
	for(i = from; i < SIZE; i ++)
		A[i] = B[i - from];
}

__device__ void  cuda_shiftLeftNum(unsigned int * A){ // shift for 32 bits in one step would be nice to consider
    char flg;
    char flc = 0;
    for (unsigned int j = 0 ; j < SIZE; j++){
    	((((unsigned int) A[j] & 0x80000000)) ? flg = 1 : flg = 0 );
    	A[j] = (A[j] << 1);
    	if(flc)
    		A[j] |= 1;
    	(flg == 1 ? flc = 1 : flc = 0);
    }
}
__device__ void  cuda_shiftRightNum(unsigned int * A){
    char flg;
    char flc = 0;
    for (unsigned char j = SIZE - 1; ; j --){
        ((((unsigned int) A[j] & 0x00000001)) ? flg = 1 : flg = 0 );
        A[j] = (A[j] >> 1);
        if(flc)
            A[j] |= 0x80000000;
        (flg == 1 ? flc = 1 : flc = 0);
        if (j == 0)
            break;
    }
}

__device__ void  cuda_addfromto(unsigned int * A, unsigned int * B, unsigned int from){
	unsigned int i;
	unsigned long int tmp;
	unsigned long int tmp_carry;
	unsigned int carry = 0;
	for(i = from; i < SIZE; i ++){
		tmp = ((unsigned long int) A[i] + B[i-from]);
		A[i] = (tmp & 0xFFFFFFFF) + carry;
		tmp_carry = ((unsigned long int) tmp & 0xFFFFFFFF00000000); 
		carry = ((unsigned long int) tmp_carry >> 32);
	}
}

__device__ void  cuda_mulNum(unsigned int * A, unsigned int * B){
    //unsigned int * tmpa = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpa[SIZE];
    //unsigned int * tmpc = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpc[SIZE];
    cuda_copyNum(tmpa, A);
    cuda_setZero(tmpc);
    unsigned long int j = 1;
    for(unsigned int k = 0; k < 32; k++){ // needed to be changed accordingly with sizeof the datatype
        for(unsigned int i = 0; i < SIZE; i++){
            if (B[i] & j){
                cuda_addfromto(tmpc, tmpa, i);
            }
        }
        j <<= 1;
        cuda_shiftLeftNum(tmpa);
    }
    cuda_copyNum(A, tmpc);
    //free(tmpa);
    //free(tmpc);
}

__device__ void  cuda_powNum(unsigned int * A){
    //unsigned int * tmpa = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpa[SIZE];
    //unsigned int * tmpc = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmpc[SIZE];
    cuda_copyNum(tmpa, A);
    cuda_setZero(tmpc); 
    unsigned long int j = 1;
    for(unsigned int k = 0; k < 32; k++){ // needed to be changed accordingly with sizeof the datatype
        for(unsigned int i = 0; i < SIZE; i++){
            if (A[i] & j){
                cuda_addfromto(tmpc, tmpa, i);
            }
        }
        j <<= 1;
        cuda_shiftLeftNum(tmpa);
    }
    cuda_copyNum(A, tmpc);
    //free(tmpa);
    //free(tmpc);
}

__device__ char cuda_zeroNum(unsigned int * num){
	unsigned int i;
	for(i = 0; i < SIZE; i ++)
		if (num[i] > 0)
			return 0;
	return 1;
}

__device__ char cuda_bigger(unsigned int * A, unsigned int * B){
	unsigned int i;
	for(i = SIZE - 1; ; i --){
		if (A[i] > B[i])
			return 1;
		if (A[i] < B[i])
			return 0;
		if (i == 0)
			break;
	}
	return 2;
}

__device__ unsigned int cuda_getHighBit(unsigned int * A){
    unsigned int bits_size = SIZE * 32;
    unsigned int l = 0;
    unsigned int tmp;
    for (long int i = SIZE - 1; i != -1; i --){
    	tmp = 0x80000000;
        while ((A[i] & tmp) == 0 && tmp != 0){
            tmp >>= 1;
            l ++;
        }
        if(tmp != 0)
            break;
    }
    l = bits_size - l;
    return l;
}


__device__ char cuda_isEven(unsigned int * A){
	if ((A[0] & 1) == 0)
		return 1;
	return 0;
}
__device__ char cuda_isOne(unsigned int * A){
	for (unsigned int i = 1; i < SIZE; i++){
		if (A[i] > 0)
			return 0;
	}
	if (A[0] == 1)
		return 1;
	return 0;
}

__device__ void  cuda_getNum(unsigned int a, unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    A[where] = 1;
    while (which){
        A[where] <<= 1;
        which --;
    }
}

__device__ unsigned int cuda_getBit(unsigned int a, unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    unsigned int pnt = ((unsigned int)1 << which);
    return ((unsigned int) A[where] & pnt); 
}
__device__ void  cuda_setBit(unsigned int a, unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    unsigned int pnt = ((unsigned int) 1 << which);
    A[where] |= pnt; 
}
__device__ void  cuda_unsBit(unsigned int a, unsigned int * A){
    unsigned int where = a / SIZE;
    unsigned int which = a % SIZE;
    unsigned int pnt = ((unsigned int) 1 << which);
    A[where] &= ~(pnt); 
}


__device__ void  cuda_divNum(unsigned int * N, unsigned int * D, unsigned int * Q){
    if (cuda_zeroNum(D) || cuda_zeroNum(N) || cuda_bigger(D, N) == 1){
        return;
    }
    //unsigned int * R = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int R[SIZE];
    cuda_setZero(Q);
    cuda_setZero(R);
    unsigned int i = SIZE*32-1;
    while (1){ // 32 is sizeof int, need to changed to sizeof
        cuda_shiftLeftNum(R);
        if (cuda_getBit(i, N)){
            R[0] |= 1;
        }else{
            R[0] &= ~((unsigned int) 1);
        }
        if(cuda_bigger(R, D) == 2 || cuda_bigger(R, D) == 1){
            cuda_subNum(R, D);
            cuda_setBit(i, Q);
        }
        if (i == 0)
            break;
        i --;
    }
    cuda_copyNum(N, R);
    //free(R);
}

__device__ void  cuda_shiftLeftNumBy(unsigned int * A, unsigned int gen){ // shift for 32 bits in one step would be nice to consider
    unsigned int howmuch = gen / 32; // needed to be changed accordingly to datatype
	unsigned int wheremc = gen % 32; // needed to be changed accordingly to datatype
	//printf("%u \n", howmuch);
	if(howmuch > 0){
		for(unsigned int k = SIZE - 1;  ; k --){
			A[k] = A[k - howmuch];
			if (k - howmuch == 0)
				break;
		}
    }
	//printf("%u \n", wheremc);
	while(wheremc){
    	cuda_shiftLeftNum(A);
    	wheremc --;
    }
    /*while(gen){
    	shiftLeftNum(A);
    	gen --;
    }*/
    /*char flg;
    char flc = 0;
    for (unsigned int j = 0 ; j < size; j++){
    	((((unsigned int) A[j] & 0x80000000)) ? flg = 1 : flg = 0 );
    	A[j] = (A[j] << 1);
    	if(flc)
    		A[j] |= 1;
    	(flg == 1 ? flc = 1 : flc = 0);
    }*/
}

__device__ void  cuda_modNum(unsigned int * A, unsigned int * B){
    if (cuda_zeroNum(A) || cuda_zeroNum(B) || cuda_bigger(B, A) == 1){
        return;
    }
    //unsigned int * tmp_b = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
    unsigned int tmp_b[SIZE];
    char gr_des;
    unsigned int dr;
    unsigned int gen;
    while( 1 ){
        cuda_copyNum(tmp_b, B);
        dr = 0;
        //gr_des = bigger(tmp_b, A);
        gen = cuda_getHighBit(A) - cuda_getHighBit(tmp_b);
        cuda_shiftLeftNumBy(tmp_b, gen);
        gr_des = cuda_bigger(tmp_b, A);
        // pokud bylo rovno ok
        // pokud bylo větší shift doprava
        if(gr_des == 1){
            cuda_shiftRightNum(tmp_b);
            dr --;
        }
        cuda_subNum(A, tmp_b); // residuo
        if(cuda_bigger(B, A)){
            break;
        }
    }
    //free(tmp_b);
}
// =======================================

/*
static long gcdl(long a, long b) {
  long r;
  while(b!=0){
    r = a % b;
    a = b;
    b = r;
		printf("**************************\n");
		printf("N:\n");
		printf("%ld \n", a);    
		printf("M:\n");
		printf("%ld \n", b);    
    getchar(); 
  }
  return a;
}*/


__device__  void cuda_gcd(unsigned int * A, unsigned int * B){
	//unsigned int * R = (unsigned int *)malloc(sizeof(unsigned int) * SIZE);
	unsigned int R[SIZE];
	while(!cuda_zeroNum(B)){
		/*printf("**************************\n");
		printf("N:\n");
		printNum(N);
		printf("M:\n");
		printNum(M);
		printf("++++++++++++++++++++++++++\n");*/
		cuda_copyNum(R, A);
		cuda_modNum(R, B);
		cuda_copyNum(A, B);
		cuda_copyNum(B, R);
		
		/*printf("**************************\n");
		printf("nm_r: \n");
		printNum(nm_r);
		printf("++++++++++++++++++++++++++\n");*/
		//getchar(); 
	}
	//free(R);
}

__device__  void cuda_fxfun(unsigned int * N, unsigned int * X, unsigned int * C){
	//setZero(Y);
	cuda_powNum(X);
	cuda_modNum(X, N);
	cuda_addNum(X, C);
	cuda_modNum(X, N);
	//copyNum(Y, X);
}

__global__ void pollardKernel(unsigned int * N, unsigned int * mem_xyc, unsigned int * result){
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int * X = &mem_xyc[3 * threadID + SIZE * 0];
	unsigned int * Y = &mem_xyc[3 * threadID + SIZE * 1];
	unsigned int * C = &mem_xyc[3 * threadID + SIZE * 2];
	unsigned int G[SIZE];
	unsigned int N_tmp[SIZE];
	unsigned int abs_mxy[SIZE];
	
	/*
	// THIS SHOULD BE PREPARED IN MEMORY
	setZero(X);
	X[0] = 7;
	setZero(C);
	C[0] = 1;
	setZero(G);
	G[0] = 1;
	*/
	cuda_copyNum(Y, X);
	cuda_fxfun(N, Y, C);
	//while (isOne(G)){
		cuda_fxfun(N, X, C);
		cuda_fxfun(N, Y, C);
		cuda_fxfun(N, Y, C);
		if(cuda_bigger(X, Y) == 1){
			cuda_copyNum(abs_mxy, X);
			cuda_subNum(abs_mxy, Y);
		}else{
			cuda_copyNum(abs_mxy, Y);
			cuda_subNum(abs_mxy, X);	
		}
		cuda_copyNum(G, abs_mxy);
		cuda_copyNum(N_tmp, N);
		cuda_gcd(G, N_tmp);
	//}
	if(! cuda_isOne(G))
		cuda_copyNum(result, G);	
}

void genNum(unsigned int * x){
	for (unsigned int i = 0; i < SIZE; i ++){
		x[i] = rand();
	}
}

void PollardRhoCu(unsigned int * N, unsigned int blocks, unsigned int threads){
	/*
The idea I've got in mind is to prepare multiple starting points
for X, Y and C
and each thread can actually count Y in place without bothering CPU
Therefore reasonable aproach would be to just prepare huge chunk of memory
for GPU to work with.
	*/
	if (isEven(N)){
		zeroNum(N);
		N[0] = 2;
		return;
	}
	unsigned int * mem_xyc = (unsigned int *) malloc(3 * blocks * threads * SIZE * sizeof(unsigned int));
	for(unsigned int i = 0; i < 3 * blocks * threads; i ++){
		genNum(mem_xyc); // X
		mem_xyc += SIZE;		
		copyNum(mem_xyc, (mem_xyc - SIZE)); // Y
		mem_xyc += SIZE;
		genNum(mem_xyc); // C
	}
	unsigned int * result = (unsigned int *) malloc(sizeof(unsigned int) * SIZE);
	setZero(result);
	
	unsigned int * gpu_xyc; 
  	hipMalloc((void **)&gpu_xyc, 3 * blocks * threads * SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_xyc, mem_xyc, 3 * blocks * threads * SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

  	unsigned int * gpu_N;
  	hipMalloc((void **)&gpu_N, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_N, N, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

  	unsigned int * gpu_result;
  	hipMalloc((void **)&gpu_result, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_result, result, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

	do{
		pollardKernel<<<blocks, threads>>>(gpu_N, gpu_xyc, gpu_result);
		hipDeviceSynchronize();
		hipMemcpy(result, gpu_result, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
	} while (zeroNum(result));

	printNum(result);
	hipFree(gpu_result);
	hipFree(gpu_N);
	hipFree(gpu_xyc);
}

void getGpuNfo(){
	// run some tests on CUDA device
    int num;
    if (hipGetDeviceCount(&num)){
    	printf("Cannot get number of CUDA devices\n");
    	return;
	}
	if (num < 1){
		printf("No CUDA devices found\n");
    	return;
	}
    hipDeviceProp_t prop;
    int   MaxDevice = -1;
    int   MaxGflops = -1;
    for (int dev=0;dev<num;dev++)
    {
        if (hipGetDeviceProperties(&prop,dev)){
        	printf("Error getting device %d properties\n",dev);
        	return;	
    	}
    	int Gflops = prop.multiProcessorCount * prop.clockRate;
        printf("CUDA Device %d: %s Gflops %f Processors %d Threads/Block %d\n",dev,prop.name,1e-6*Gflops,prop.multiProcessorCount,prop.maxThreadsPerBlock);
        if(Gflops > MaxGflops)
        {
            MaxGflops = Gflops;
            MaxDevice = dev;
        }
    }
    
    //  Print and set device
    if (hipGetDeviceProperties(&prop,MaxDevice)){
    	printf("Error getting device %d properties\n", MaxDevice);
    	return;
    }
    printf("Fastest CUDA Device %d: %s\n",MaxDevice,prop.name);
    hipSetDevice(MaxDevice);
    //  Return max thread count
	printf("maxThreadsPerBlock %d \n", prop.maxThreadsPerBlock);
	return;
}

int main(int argc, char **argv) {
	srand (time(NULL));
	getGpuNfo();
	unsigned int N[SIZE];
	setZero(N);
    //N[8] = 1;
    //N[8] = 1;
    //N[0] = 1;
    N[1] = 0x00000111;
    N[0] = 0x10010009;
    PollardRhoCu(N, 1, 1);
	printf("Results \n");
    printNum(N);
	printf("********************\n");
	/*unsigned int X[32];
	unsigned int C[32];
	unsigned int X[32];
	unsigned int Y[32];
	unsigned int C[32];
	unsigned int N[32];*/
	//void fxfun(unsigned int * N, unsigned int * X, unsigned int * C, unsigned int * Y)
	//setZero(N, 32);	
	//setZero(R, 32);
	//N[0] = 0x00000009;
	//0xF00000007530eca9
    /*N[3] = 0x8ebaf1b9;
    N[2] = 0xfebaf1ba;
    N[1] = 0xfeba9876;
    N[0] = 0x7530eca9;*/
    /*0 x 1 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000001*/
    /*N[8] = 1;
    N[0] = 1;
    PollardRho(N, R, 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(R, 32);
	printf("********************\n");
	*//*	
	setZero(A, 32);
	setZero(C, 32);
	setZero(N, 32);	
	setZero(X, 32);	
	setZero(Y, 32);	
	//N[1] = 0xF0010999;
    N[0] = 0x12345678;
    C[1] = 0x00000001;
	X[15] = 0x00000001;
	X[0] = 0x10000002;
	fxfun(N, X, C, Y , 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(X, 32);
    printNum(C, 32);
	printNum(Y, 32);	*/
	/*unsigned int A[32];
	unsigned int B[32];
	printf("********************\n"); 	
	setZero(A, 32);
	setZero(B, 32);
	//1B6 9B4B ACD0 5F15	
	//A[1] = 0x01B69B4B;
    //A[0] = 0x00000000;
	A[31] = 0x80000999; // to keep
    A[0] = 0x10000002; // to keep
	//A[0] = 0x80000003;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	//B[1] = 0x00000001;
	//B[0] = 0x00000000;
	//B[1] = 0x80000999; // to keep
	B[0] = 0x10001010; // to keep
	//gcd(A, B, 32);
    modNum(A, B, 32);
    printf("Results \n");
    printNum(A, 32);*/
	
	//printNum(B, 32);
/*
	printf("********************\n");
    unsigned int A[2*32];
	unsigned int B[2*32];
	setZero(A, 2*32);
	setZero(B, 2*32);
	A[0] = 1;    
	copyFromTo(B, A, 32, 2*32);
	printNum(B, 2*32);
*/
   /* printf("********************\n");
    unsigned int N[32];
	setZero(N, 32);
    unsigned int D[32];
	setZero(D, 32);
    unsigned int Q[32];
	setZero(Q, 32);
    
    //N[1] = 0x00000111;
    N[0] = 0x00000008;
    //D[1] = 0x00000111;
    D[0] = 0x00000002;
    
    divNum(N, D, Q, 32);
    printf("Results\n");
    printNum(N, 32);
    printNum(D, 32);
    printNum(Q, 32);
    printf("********************\n");
	return 0;*/
	/*
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[1] = 0xF0010999;
    A[0] = 0x12345678;
    B[1] = 0xF0000000;
	B[0] = 0x7530eca9;
	gcd(A, B, 32);
    //modNum(A, B, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(R, 32);
	printf("********************\n");*/
	/*setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	//1B6 9B4B ACD0 5F15
	A[0] = 0x00000008;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	B[0] = 0x00000008;
	divNum(A, B, D, R, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[0] = 0x89;
    //DB4 DA5F 7EF4 12B1
	B[0] = 0x21;
	gcd(A, B, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);*/
	//printf("gcdl %ld\n",gcdl(0x00000005, 0x00000002));
	return 0;
}