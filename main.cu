#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h> 
#include "BigNum/BigNum.h"
#include "BigNum/BigNum.cuh"
#define SIZE ((9))

//

/*
static long gcdl(long a, long b) {
  long r;
  while(b!=0){
    r = a % b;
    a = b;
    b = r;
		printf("**************************\n");
		printf("N:\n");
		printf("%ld \n", a);    
		printf("M:\n");
		printf("%ld \n", b);    
    getchar(); 
  }
  return a;
}*/


void getGpuNfo(){
	// run some tests on CUDA device
    int num;
    if (hipGetDeviceCount(&num)){
    	printf("Cannot get number of CUDA devices\n");
    	return;
	}
	if (num < 1){
		printf("No CUDA devices found\n");
    	return;
	}
    hipDeviceProp_t prop;
    int   MaxDevice = -1;
    int   MaxGflops = -1;
    for (int dev=0;dev<num;dev++)
    {
        if (hipGetDeviceProperties(&prop,dev)){
        	printf("Error getting device %d properties\n",dev);
        	return;	
    	}
    	int Gflops = prop.multiProcessorCount * prop.clockRate;
        printf("CUDA Device %d: %s Gflops %f Processors %d Threads/Block %d\n",dev,prop.name,1e-6*Gflops,prop.multiProcessorCount,prop.maxThreadsPerBlock);
        if(Gflops > MaxGflops)
        {
            MaxGflops = Gflops;
            MaxDevice = dev;
        }
    }
    
    //  Print and set device
    if (hipGetDeviceProperties(&prop,MaxDevice)){
    	printf("Error getting device %d properties\n", MaxDevice);
    	return;
    }
    printf("Fastest CUDA Device %d: %s\n",MaxDevice,prop.name);
    // hipSetDevice(MaxDevice);    
    hipSetDevice(0);
    //  Return max thread count
	printf("maxThreadsPerBlock %d \n", prop.maxThreadsPerBlock);
	return;
}


void genNum(unsigned int * x){
	setZero(x);
	x[0] = 0x07;
}

void genC(unsigned int * x, unsigned int * counter){
	setZero(x);
	copyNum(x, counter);
	addOne(counter);
}

void printMat(unsigned int * mem_xyc, unsigned int blocks, unsigned int threads){
	for(unsigned int i = 0; i < 3 * blocks * threads; i += 3){
		printNum ( mem_xyc + SIZE * ( i     )); // X
		printNum ( mem_xyc + SIZE * ( i + 1 )); // Y
		printNum ( mem_xyc + SIZE * ( i + 2 )); // C 
		printf("===================================================== \n");
	}
}

void PollardRhoCu(unsigned int * N, unsigned int blocks, unsigned int threads){
	/*
The idea I've got in mind is to prepare multiple starting points
for X, Y and C
and each thread can actually count Y in place without bothering CPU
Therefore reasonable aproach would be to just prepare huge chunk of memory
for GPU to work with.
	*/
	if (isEven(N)){
		zeroNum(N);
		N[0] = 2;
		return;
	}
	unsigned int  counter[SIZE];
	setZero(counter);
	counter[0] = 0x01;

	unsigned int * mem_xyc = (unsigned int *) malloc(3 * blocks * threads * SIZE * sizeof(unsigned int));
	for(unsigned int i = 0; i < 3 * blocks * threads; i += 3){
		genNum   ( mem_xyc + SIZE * ( i     )); // X
		copyNum  ( mem_xyc + SIZE * ( i + 1 ), mem_xyc + SIZE * i); // Y
		genC     ( mem_xyc + SIZE * ( i + 2 ), counter ); // C 
		fxfun    (N, mem_xyc + SIZE * ( i + 1 ), mem_xyc + SIZE * ( i + 2 ));
	}

	unsigned int * result = (unsigned int *) malloc(sizeof(unsigned int) * SIZE);
	setZero(result);
	
	unsigned int * gpu_xyc; 
  	hipMalloc((void **)&gpu_xyc, 3 * blocks * threads * SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_xyc, mem_xyc, 3 * blocks * threads * SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

	//printMat(mem_xyc, blocks, threads);

  	unsigned int * gpu_N;
  	hipMalloc((void **)&gpu_N, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_N, N, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
  	unsigned int * gpu_result;
  	hipMalloc((void **)&gpu_result, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_result, result, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

/*
	unsigned int * cu_dbgs; 
  	hipMalloc((void **)&cu_dbgs, sizeof(unsigned int));
	unsigned int * ma_dbgs = (unsigned int *) malloc(sizeof(unsigned int)); 
*/
	unsigned int it = 0;
	printf("Running Kernel\n");
	do{
		//pollardKernel<<<blocks, threads>>>(gpu_N, gpu_xyc, gpu_result, cu_dbgs);
		pollardKernel<<<blocks, threads>>>(gpu_N, gpu_xyc, gpu_result);
		hipDeviceSynchronize();
		//hipMemcpy(ma_dbgs, cu_dbgs, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);		
		//break;
		
		//hipMemcpy(mem_xyc, gpu_xyc, 3 * blocks * threads * SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
		//printMat(mem_xyc, blocks, threads);
		//printf("++++++++++++++++++++++++++++++++++++++++++\n");
		hipMemcpy(result, gpu_result, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
		//sleep(1);
		if(it % 1000 == 0 && it != 0)
			printf("%u\n", it);
		it ++;
	} while (zeroNum(result));

	copyNum(N, result);
	hipFree(gpu_result);
	hipFree(gpu_N);
	hipFree(gpu_xyc);
}


int main(int argc, char **argv) {
	hipSetDevice(0);
	//getGpuNfo();
	unsigned int N[SIZE];
	setZero(N);
    //N[8] = 1;
    //N[8] = 1;
    //N[0] = 1;
    //N[1] = 0x00000111;
    //N[0] = 0x10010009;
    //N[0] = 0x00044d69;
    //123432322333420120051
    //N[0] = 0x20120051;
    //N[1] = 0x23223334;
    //N[3] = 0x00012343;
    // cb3b6f92b9d54fd
    //N[1] = 0x0cb3b6f9;
    //N[0] = 0x2b9d54fd;
    //a8bc39b45
    //N[1] = 0x0000000a;
    //N[0] = 0x8bc39b45;
    //
    //N[0] = 0x00000121;
    //fd42d4eb2c4b7b1
    N[1] = 0x0fd42d4e;
    N[0] = 0xb2c4b7b1;
    

    PollardRhoCu(N, 128, 128);
	printf("Results \n");
    printNum(N);
	printf("********************\n");
	/*unsigned int X[32];
	unsigned int C[32];
	unsigned int X[32];
	unsigned int Y[32];
	unsigned int C[32];
	unsigned int N[32];*/
	//void fxfun(unsigned int * N, unsigned int * X, unsigned int * C, unsigned int * Y)
	//setZero(N, 32);	
	//setZero(R, 32);
	//N[0] = 0x00000009;
	//0xF00000007530eca9
    /*N[3] = 0x8ebaf1b9;
    N[2] = 0xfebaf1ba;
    N[1] = 0xfeba9876;
    N[0] = 0x7530eca9;*/
    /*0 x 1 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000001*/
    /*N[8] = 1;
    N[0] = 1;
    PollardRho(N, R, 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(R, 32);
	printf("********************\n");
	*//*	
	setZero(A, 32);
	setZero(C, 32);
	setZero(N, 32);	
	setZero(X, 32);	
	setZero(Y, 32);	
	//N[1] = 0xF0010999;
    N[0] = 0x12345678;
    C[1] = 0x00000001;
	X[15] = 0x00000001;
	X[0] = 0x10000002;
	fxfun(N, X, C, Y , 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(X, 32);
    printNum(C, 32);
	printNum(Y, 32);	*/
	/*unsigned int A[32];
	unsigned int B[32];
	printf("********************\n"); 	
	setZero(A, 32);
	setZero(B, 32);
	//1B6 9B4B ACD0 5F15	
	//A[1] = 0x01B69B4B;
    //A[0] = 0x00000000;
	A[31] = 0x80000999; // to keep
    A[0] = 0x10000002; // to keep
	//A[0] = 0x80000003;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	//B[1] = 0x00000001;
	//B[0] = 0x00000000;
	//B[1] = 0x80000999; // to keep
	B[0] = 0x10001010; // to keep
	//gcd(A, B, 32);
    modNum(A, B, 32);
    printf("Results \n");
    printNum(A, 32);*/
	
	//printNum(B, 32);
/*
	printf("********************\n");
    unsigned int A[2*32];
	unsigned int B[2*32];
	setZero(A, 2*32);
	setZero(B, 2*32);
	A[0] = 1;    
	copyFromTo(B, A, 32, 2*32);
	printNum(B, 2*32);
*/
   /* printf("********************\n");
    unsigned int N[32];
	setZero(N, 32);
    unsigned int D[32];
	setZero(D, 32);
    unsigned int Q[32];
	setZero(Q, 32);
    
    //N[1] = 0x00000111;
    N[0] = 0x00000008;
    //D[1] = 0x00000111;
    D[0] = 0x00000002;
    
    divNum(N, D, Q, 32);
    printf("Results\n");
    printNum(N, 32);
    printNum(D, 32);
    printNum(Q, 32);
    printf("********************\n");
	return 0;*/
	/*
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[1] = 0xF0010999;
    A[0] = 0x12345678;
    B[1] = 0xF0000000;
	B[0] = 0x7530eca9;
	gcd(A, B, 32);
    //modNum(A, B, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(R, 32);
	printf("********************\n");*/
	/*setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	//1B6 9B4B ACD0 5F15
	A[0] = 0x00000008;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	B[0] = 0x00000008;
	divNum(A, B, D, R, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[0] = 0x89;
    //DB4 DA5F 7EF4 12B1
	B[0] = 0x21;
	gcd(A, B, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);*/
	//printf("gcdl %ld\n",gcdl(0x00000005, 0x00000002));
	return 0;
}
