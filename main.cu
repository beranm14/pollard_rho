#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <time.h> 
#include "BigNum/BigNum.h"
#include "BigNum/BigNum.cuh"
#define SIZE ((8))

/*
static long gcdl(long a, long b) {
  long r;
  while(b!=0){
    r = a % b;
    a = b;
    b = r;
		printf("**************************\n");
		printf("N:\n");
		printf("%ld \n", a);    
		printf("M:\n");
		printf("%ld \n", b);    
    getchar(); 
  }
  return a;
}*/



void genNum(unsigned int * x, unsigned int * N){
	for (unsigned int i = 0; i < SIZE; i ++){
		x[i] = rand();
	}
	divNum(x, N);
}

void PollardRhoCu(unsigned int * N, unsigned int blocks, unsigned int threads){
	/*
The idea I've got in mind is to prepare multiple starting points
for X, Y and C
and each thread can actually count Y in place without bothering CPU
Therefore reasonable aproach would be to just prepare huge chunk of memory
for GPU to work with.
	*/
	if (isEven(N)){
		zeroNum(N);
		N[0] = 2;
		return;
	}
	unsigned int * mem_xyc = (unsigned int *) malloc(3 * blocks * threads * SIZE * sizeof(unsigned int));
	for(unsigned int i = 0; i < 3 * blocks * threads; i ++){
		genNum(mem_xyc, N); // X
		copyNum(mem_xyc, (mem_xyc + 1 * SIZE)); // Y
		genNum(mem_xyc + 2 * SIZE, N); // C
	}
	unsigned int * result = (unsigned int *) malloc(sizeof(unsigned int) * SIZE);
	setZero(result);
	
	unsigned int * gpu_xyc; 
  	hipMalloc((void **)&gpu_xyc, 3 * blocks * threads * SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_xyc, mem_xyc, 3 * blocks * threads * SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

  	unsigned int * gpu_N;
  	hipMalloc((void **)&gpu_N, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_N, N, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

  	unsigned int * gpu_result;
  	hipMalloc((void **)&gpu_result, SIZE * sizeof(unsigned int));
	hipMemcpy(gpu_result, result, SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);

	do{
		pollardKernel<<<blocks, threads>>>(gpu_N, gpu_xyc, gpu_result);
		hipDeviceSynchronize();
		hipMemcpy(result, gpu_result, SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
	} while (zeroNum(result));

	printNum(result);
	hipFree(gpu_result);
	hipFree(gpu_N);
	hipFree(gpu_xyc);
}

void getGpuNfo(){
	// run some tests on CUDA device
    int num;
    if (hipGetDeviceCount(&num)){
    	printf("Cannot get number of CUDA devices\n");
    	return;
	}
	if (num < 1){
		printf("No CUDA devices found\n");
    	return;
	}
    hipDeviceProp_t prop;
    int   MaxDevice = -1;
    int   MaxGflops = -1;
    for (int dev=0;dev<num;dev++)
    {
        if (hipGetDeviceProperties(&prop,dev)){
        	printf("Error getting device %d properties\n",dev);
        	return;	
    	}
    	int Gflops = prop.multiProcessorCount * prop.clockRate;
        printf("CUDA Device %d: %s Gflops %f Processors %d Threads/Block %d\n",dev,prop.name,1e-6*Gflops,prop.multiProcessorCount,prop.maxThreadsPerBlock);
        if(Gflops > MaxGflops)
        {
            MaxGflops = Gflops;
            MaxDevice = dev;
        }
    }
    
    //  Print and set device
    if (hipGetDeviceProperties(&prop,MaxDevice)){
    	printf("Error getting device %d properties\n", MaxDevice);
    	return;
    }
    printf("Fastest CUDA Device %d: %s\n",MaxDevice,prop.name);
    hipSetDevice(MaxDevice);
    //  Return max thread count
	printf("maxThreadsPerBlock %d \n", prop.maxThreadsPerBlock);
	return;
}

int main(int argc, char **argv) {
	srand (time(NULL));
	getGpuNfo();
	unsigned int N[SIZE];
	setZero(N);
    //N[8] = 1;
    //N[8] = 1;
    //N[0] = 1;
    //N[1] = 0x00000111;
    //N[0] = 0x10010009;
    //123432322333420120051
    N[0] = 0x20120051;
    N[1] = 0x23223334;
    N[3] = 0x00012343;
    PollardRhoCu(N, 1024, 1024);
	printf("Results \n");
    printNum(N);
	printf("********************\n");
	/*unsigned int X[32];
	unsigned int C[32];
	unsigned int X[32];
	unsigned int Y[32];
	unsigned int C[32];
	unsigned int N[32];*/
	//void fxfun(unsigned int * N, unsigned int * X, unsigned int * C, unsigned int * Y)
	//setZero(N, 32);	
	//setZero(R, 32);
	//N[0] = 0x00000009;
	//0xF00000007530eca9
    /*N[3] = 0x8ebaf1b9;
    N[2] = 0xfebaf1ba;
    N[1] = 0xfeba9876;
    N[0] = 0x7530eca9;*/
    /*0 x 1 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000001*/
    /*N[8] = 1;
    N[0] = 1;
    PollardRho(N, R, 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(R, 32);
	printf("********************\n");
	*//*	
	setZero(A, 32);
	setZero(C, 32);
	setZero(N, 32);	
	setZero(X, 32);	
	setZero(Y, 32);	
	//N[1] = 0xF0010999;
    N[0] = 0x12345678;
    C[1] = 0x00000001;
	X[15] = 0x00000001;
	X[0] = 0x10000002;
	fxfun(N, X, C, Y , 32);
	printf("Results \n");
    printNum(N, 32);
	printNum(X, 32);
    printNum(C, 32);
	printNum(Y, 32);	*/
	/*unsigned int A[32];
	unsigned int B[32];
	printf("********************\n"); 	
	setZero(A, 32);
	setZero(B, 32);
	//1B6 9B4B ACD0 5F15	
	//A[1] = 0x01B69B4B;
    //A[0] = 0x00000000;
	A[31] = 0x80000999; // to keep
    A[0] = 0x10000002; // to keep
	//A[0] = 0x80000003;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	//B[1] = 0x00000001;
	//B[0] = 0x00000000;
	//B[1] = 0x80000999; // to keep
	B[0] = 0x10001010; // to keep
	//gcd(A, B, 32);
    modNum(A, B, 32);
    printf("Results \n");
    printNum(A, 32);*/
	
	//printNum(B, 32);
/*
	printf("********************\n");
    unsigned int A[2*32];
	unsigned int B[2*32];
	setZero(A, 2*32);
	setZero(B, 2*32);
	A[0] = 1;    
	copyFromTo(B, A, 32, 2*32);
	printNum(B, 2*32);
*/
   /* printf("********************\n");
    unsigned int N[32];
	setZero(N, 32);
    unsigned int D[32];
	setZero(D, 32);
    unsigned int Q[32];
	setZero(Q, 32);
    
    //N[1] = 0x00000111;
    N[0] = 0x00000008;
    //D[1] = 0x00000111;
    D[0] = 0x00000002;
    
    divNum(N, D, Q, 32);
    printf("Results\n");
    printNum(N, 32);
    printNum(D, 32);
    printNum(Q, 32);
    printf("********************\n");
	return 0;*/
	/*
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[1] = 0xF0010999;
    A[0] = 0x12345678;
    B[1] = 0xF0000000;
	B[0] = 0x7530eca9;
	gcd(A, B, 32);
    //modNum(A, B, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(R, 32);
	printf("********************\n");*/
	/*setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	//1B6 9B4B ACD0 5F15
	A[0] = 0x00000008;
	//DB4 DA5F 7EF4 12B1
	//B[1] = 0x0DB4DA5F;
	B[0] = 0x00000008;
	divNum(A, B, D, R, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	printf("********************\n");
	setZero(A, 32);
	setZero(B, 32);
	setZero(D, 32);
	setZero(R, 32);	
	A[0] = 0x89;
    //DB4 DA5F 7EF4 12B1
	B[0] = 0x21;
	gcd(A, B, 32);
    //divNum(A, B, D, R, 32);
    printf("Results \n");
    printNum(A, 32);
	printNum(B, 32);
    printNum(D, 32);
	printNum(R, 32);*/
	//printf("gcdl %ld\n",gcdl(0x00000005, 0x00000002));
	return 0;
}